#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>

#include "bloom.h"
#include "murmuda3.h"

__global__
void cuda_add(uint32_t* cuda_bit_vector, int num_bits, uint32_t* cuda_seeds,
              int num_seeds, const void* cuda_key, int len) {
    // For now lets pretend this is just called for one key
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Allocate memory on device from kernel for output of hash. This is init
    // by the decorator
    extern __shared__ uint32_t out[];
    uint32_t bit_index;

    // Hash them in parallel
    for (int k = index; k < num_seeds; k+= stride) {
        _Murmur3_helper(cuda_key, len, cuda_seeds[k], &(out[k]));

        // Use cuda atomic functions to guarentee it is flipped
        bit_index = out[k] % num_bits;
        atomicOr(&(cuda_bit_vector[bit_index / 32]),
                 (uint32_t) 1 << (bit_index % 32));
    }
}

__global__
void cuda_test(uint32_t* cuda_bit_vector, int num_bits, uint32_t* cuda_seeds,
               int num_seeds, const void* cuda_key, int len, bool * bool_out) {
    // For now lets pretend this is just called for one key
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Allocate memory on device from kernel for output of hash. This is init
    // by the decorator
    extern __shared__ bool test_vals[];
    uint32_t out, bit_index;

    // Hash them in parallel
    for (int k = index; k < num_seeds; k+= stride) {
        _Murmur3_helper(cuda_key, len, cuda_seeds[k], &out);
        bit_index = out % num_bits;
        test_vals[k] = (cuda_bit_vector[bit_index / 32] & (1 << (bit_index % 32)));
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        *bool_out = true;
        for (int i = 0; i < num_seeds; i++) {
            if (!test_vals[i]) {
                *bool_out = false;
                break;
            }
        }
    }
}


BloomFilter::BloomFilter(int n_bits, int n_seeds) {
    num_bits = n_bits;

    num_int = (num_bits + (sizeof(uint32_t) - 1)) / sizeof(uint32_t);
    bit_vector =  new uint32_t[num_int];
    std::fill(bit_vector, bit_vector+num_int, 0);

    // Allocate the bit vector on the device
    hipMalloc(&cuda_bit_vector, num_int * sizeof(bit_vector[0]));
    hipMemcpy(cuda_bit_vector, bit_vector, num_int * sizeof(bit_vector[0]),
               hipMemcpyHostToDevice);


    num_seeds = n_seeds;
    seeds = new uint32_t[num_seeds];
    for (int i = 0; i < num_seeds; i++) {
        seeds[i] = i;
    }

    hipMalloc(&cuda_seeds, num_seeds * sizeof(uint32_t));
    hipMemcpy(cuda_seeds, seeds, num_seeds * sizeof(uint32_t),
               hipMemcpyHostToDevice);
}


void BloomFilter::add(const void * key, int len) {
    void * cuda_key;
    hipMalloc(&cuda_key, len);
    hipMemcpy(cuda_key, key, len, hipMemcpyHostToDevice);

    int blockSize = num_seeds;
    int numBlocks = 1;

    cuda_add<<<numBlocks,
               blockSize,
               num_seeds * sizeof(uint32_t)>>>(cuda_bit_vector, num_bits,
                                               cuda_seeds, num_seeds,
                                               cuda_key, len);

    hipDeviceSynchronize();

    hipFree(cuda_key);
}

void BloomFilter::sync() {
    hipMemcpy(bit_vector, cuda_bit_vector, num_int * sizeof(bit_vector[0]),
               hipMemcpyDeviceToHost);
}

bool BloomFilter::test(const void * key, int len) {
    bool result;
    bool * cuda_result;
    hipMalloc(&cuda_result, sizeof(bool));

    void * cuda_key;
    hipMalloc(&cuda_key, len);
    hipMemcpy(cuda_key, key, len, hipMemcpyHostToDevice);

    int blockSize = num_seeds;
    int numBlocks = 1;

    cuda_test<<<numBlocks,
                blockSize,
                num_seeds * sizeof(uint32_t)>>>(cuda_bit_vector, num_bits,
                                                cuda_seeds, num_seeds,
                                                cuda_key, len, cuda_result);

    hipDeviceSynchronize();

    hipMemcpy(&result, cuda_result, sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(cuda_key);
    hipFree(cuda_result);

    return result;
}
